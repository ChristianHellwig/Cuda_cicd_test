#include "hip/hip_runtime.h"
#include <stdio.h>
#include "xcuda.cuh"

// __global__ functions, or "kernels", execute on the device
__global__ void helloKernel(int* a)
{
	printf("Hello, world from the device %d!\n", *a);
	atomicAdd(a, 1);
	printf("Hello, world from the device %d!\n", *a);
}

int callKernel(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  int a = 10;
  int* d_a;
  const auto cudaStatus = hipMalloc((void**)&d_a, sizeof(int));

  if (cudaStatus != hipSuccess) {
	  printf("Err!\n");
	  return -1;
  }

  hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);

  // launch a kernel with a single thread to greet from the device
  helloKernel<<<1,1>>>(d_a);


  hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
  printf("Res: %d\n", a);

  // wait for the device to finish so that we see the message
  hipDeviceSynchronize();

  return 0;
}
 