
#include <hip/hip_runtime.h>
#include <stdio.h>

// __global__ functions, or "kernels", execute on the device
__global__ void helloKernel(void)
{
  printf("Hello, world from the device!\n");
}

int callKernel(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  helloKernel<<<1,1>>>();

  // wait for the device to finish so that we see the message
  hipDeviceSynchronize();

  return 0;
}
 